#include "hip/hip_runtime.h"
#include <stdio.h>
const int threshold=400;
const int m=100;
__global__ void selection_sort(int *a, int left, int right)
{
	int temp;
	 for(int i=left;i<right;i++)
    	for(int j=i+1;j<=right;j++)
    		if(a[i]>a[j])
    		{
    			temp=a[i];
    			a[i]=a[j];
    			a[j]=temp;
    		}
}

__global__ void partition(int *a,int left,int right,int pivot,int *al,int *ah)
{
	int l,h;
	int diff=(right-left+1)/m;
	int k1=threadIdx.x*diff+left;
	int k2=k1+diff-1;
	if(threadIdx.x==m-1)
		k2=right;
	l=h=k1;
	for(int i=k1;i<=k2;i++)
		{
			al[i]=ah[i]=-999;
		}
	for(int i=k1;i<=k2;i++)
	{
		if(a[i]<pivot)
		{
			al[l++]=a[i];
		}
		else
		{
			if(a[i]>pivot)
			{
				ah[h++]=a[i];
			}
		}
	}
}

void quicksort(int *a, const int left, const int right)
{
	if (right-left <= threshold)
    {
    	int *ad;
    	hipMalloc((void **)&ad,(right-left+1)*sizeof(int));
    	hipMemcpy(ad,a,(right-left+1)*sizeof(int),hipMemcpyHostToDevice);
        selection_sort<<<1,1>>>(ad, left, right);
        hipMemcpy(a,ad,(right-left+1)*sizeof(int),hipMemcpyDeviceToHost);
        return;
    }
    int pivot=a[left];
    int *al,*ah;
    int *ad;
    hipMalloc((void **)&ad,(right-left+1)*sizeof(int));
    hipMalloc((void **)&al,(right-left+1)*sizeof(int));
    hipMalloc((void **)&ah,(right-left+1)*sizeof(int));
    hipMemcpy(ad,a,(right-left+1)*sizeof(int),hipMemcpyHostToDevice);
    partition<<<1,m>>>(ad,left,right,pivot,al,ah);
    int al_h[right-left+1],ah_h[right-left+1];
        hipMemcpy(al_h,al,(right-left+1)*sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(ah_h,ah,(right-left+1)*sizeof(int),hipMemcpyDeviceToHost);
    int i=0,k=0;
    while(i<right-left+1)
    {
    	while(al_h[i]==-999 && i<right-left+1)
    			i++;
    	while(al_h[i]!=-999 && i<right-left+1)
    	{
    		al_h[k++]=al_h[i++];
    	}
    }
    quicksort(al_h,0,k-1);
    int p=left;
    int x=0;

        while(x<k)
        {
        	a[p++]=al_h[x++];

        }
        a[p]=pivot;
    i=0;
    k=0;
    while(i<right-left+1)
    {
      	while(ah_h[i]==-999 && i<right-left+1)
    		i++;
       	while(ah_h[i]!=-999 && i<right-left+1)
       	{
       		ah_h[k++]=ah_h[i++];
       	}
    }
    quicksort(ah_h,0,k-1);
    i=0;
    p++;
        while(i<k)
        {
        	a[p++]=ah_h[i++];
        }
}
int main()
{
	int n = 4000;
	int a[n];
	time_t t;
    srand((unsigned)time(&t));
    int x,flag;
    for (unsigned i = 0 ; i < n ; i++)
    {
    	x=rand()%n;
    	flag=0;
    	for(int j=0;j<i;j++)
    	{
    		if(a[j]==x)
    		{
    			i--;
    			flag=1;
    			break;
    		}
    	}
    	if(flag==0)
    		a[i]=x;
    }
    printf("\n\n original array\n");
    for(int i=0;i<n;i++)
    	printf("%d\t ",a[i]);
    quicksort(a,0,n-1);
    printf("\n\n after sorting\n");
    for(int i=0;i<n;i++)
        	printf("%d\t ",a[i]);
}
