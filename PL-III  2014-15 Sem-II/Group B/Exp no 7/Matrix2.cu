//Matrix Multiplication Program Using threading

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
int Width=32;	//Width of matrix

int hTile_Width=300;
__device__ int dTile_Width=32; //Declare device copy for tile width

int CompareMatrix(int *M1,int *M2,int row,int col)
{		for(int i=0;i<(row*col);i++)
		{	if(M1[i]!=M2[i])
				return i;	
		}
		return -1;
		
}

/*Function to display Matrix elements
--------------------------------------------
Arguments
1-Matrix Pointer
2-Total elements size
--------------------------------------------
*/


void display_matrix(int *ptr,int m, int n)
{
	for(int i=0;i<m*n;i++)
	{	if(i%(n)==0){
			printf("\n");
		}
		printf("%d ",ptr[i]);
	}
}

/*Kernel function
---------------------------------------------
Arguments:
1-1st Matrix pointer
2-2nd Matrix pointer
3-Resultant matrix pointer
4-Width of square matrix
	This function calculates dot product of row and column for each thread
	__global__ qualifier indicates function can be called from host code and runs on device(GPU)
---------------------------------------------
*/
__global__ void MatrixMulKernel(int *Md,int *Nd,int *Pd,int w)
{
	int row=blockIdx.y*dTile_Width+ threadIdx.y;		//Get thread X position in a 2D block of threads
   int col=blockIdx.x*dTile_Width+ threadIdx.x;		//Get thread Y position in a 2D block of threads
   int Pvalue=0;
	 if(row<w && col<w){
    	for(int k=0;k<w;++k)	//
    	{
        Pvalue+=Md[row*w+k]*Nd[k*w+col];		//Calculate sum of dot product of elements
    	}
    	Pd[row*w+col]=Pvalue;
	}	
}

void NormalMultiplication(int *M,int *N,int *P,int w)
{

	for(int i=0;i<w;++i){
		for(int j=0;j<w;j++){
			int sum=0;
			for(int k=0;k<w;k++){
				int a=M[i*w+k];
				int b=N[k*w+j];
				sum+=a*b;
			}
			P[i*w+j]=sum;
		}
	}

}

/*
-----------------------------------------------
Arguments
1-1st Matrix pointer
2-2nd Matrix pointer
3-Resultant matrix pointer
4-Width of square matrix
	This function copy matrix data from host memory to device global memory and invoke kernel
-----------------------------------------------
 */
void MatrixMultiplication(int *M,int *N,int *P,int w)
{	int *Md,*Nd,*Pd;		//Matrix Pointer on device memoryi.e GPU
	int size=w*w*sizeof(int*);
	int x;

	hipMalloc((void**)&Md,size);		//Allocate memory on device global memory
	hipMemcpy(Md,M,size,hipMemcpyHostToDevice);	//Copy matrix data from host to device memory
	hipMalloc((void**)&Nd,size);
	hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);
	hipMalloc((void**)&Pd,size);
	

			if(w%hTile_Width!=0)			//
			{
				x=w/hTile_Width+1;
			}
			else
			{
				x=w/hTile_Width;
			}
		
		dim3 dimGrid(x,x);									//Variable for threads arrangement in a block.
		dim3 dimBlock(hTile_Width,hTile_Width);		//Variable for blocks arrangement in a grid.	

	MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,w);	//Kernel invocation with grid and block specification in angle brackets

	hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);	//Copy resultant matrix from device to host
	//Free device memory
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);

}

int main()
{
    int mat_size=Width*Width*sizeof(int*);	//Calculate memory size required for float matrix
    int tot_elements=Width*Width;
    int *M,*N,*P,*ptr,*P_CPU;	// Host matrix pointers
	int a=0;
	int i=0;

	M=(int*)malloc(mat_size);		//Allocate memory on host for matrix
	N=(int*)malloc(mat_size);
	P=(int*)malloc(mat_size);
	P_CPU=(int*)malloc(mat_size);
	ptr=M;
	printf("\nGenarating random elements for matrix");
	for(i=0;i<tot_elements;i++)
	{	a=(rand()%10);		//Generates random no. in 0 to 10 range
		*ptr=a;
		ptr++;
	}
	ptr=N;
	for(i=0;i<tot_elements;i++)
	{
		a=(rand()%10);
		*ptr=a;
		ptr++;
	}
	printf("\nMatrix generated");
	MatrixMultiplication(M,N,P,Width);
	printf("\nDone with GPU");
	display_matrix(N,Width,Width);
	printf("\nMatrix Multiplication (GPU) is :");
	display_matrix(P,Width,Width);
	NormalMultiplication(M,N,P_CPU,Width);
	printf("\nDone with CPU");	
	printf("\n\nMatrix Multiplication is :");
	display_matrix(P_CPU,Width,Width);

	if(CompareMatrix(P,P_CPU,Width,Width)==-1)
		printf("\n\nBoth matrix are same\n");
	else
	{	printf("\n\nBoth matrix are not same\n\n");
		//printf("%d",CompareMatrix(P,P_CPU,Width,Width));
	}
}
