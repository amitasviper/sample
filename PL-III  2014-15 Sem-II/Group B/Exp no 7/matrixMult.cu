// Multiply two matrices A * B = C
 
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <iostream>
using namespace std;
#include <time.h>
#include<hip/hip_runtime.h>
// includes CUDA Runtime


// maybe you need also helpers

#define BLOCK_SIZE 30
 
#define WA 30// Matrix A width
#define HA 30 // Matrix A height
#define WB 30 // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width
#define HC HA  // Matrix C height


__global__ void
matrixMul( float* C, float* A, float* B, int wA, int wB)
{
 
   // 2D Thread ID
   int tx = threadIdx.x;
   int ty = threadIdx.y;
 
   // value stores the element that is 
   // computed by the thread
   float value = 0;
   for (int i = 0; i < wA; ++i)
   {
      float elementA = A[ty * wA + i];
      float elementB = B[i * wB + tx];
      value += elementA * elementB;
   }
 
   // Write the matrix to device memory each 
   // thread writes one element
   C[ty * wA + tx] = value;
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}
 
 
int main(int argc, char** argv){
 
    // set seed for rand()
    srand(2006);
 
    // 1. allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
 
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);
 
    // 2. initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
  
    // 3. print out A and B
    printf("\n\nMatrix A\n");
    for(int i = 0; i < size_A; i++)
    {
       printf("%f ", h_A[i]);
       if(((i + 1) % WA) == 0)
          printf("\n");
    }
 
    printf("\n\nMatrix B\n");
    for(int i = 0; i < size_B; i++)
    {
       printf("%f ", h_B[i]);
       if(((i + 1) % WB) == 0)
          printf("\n");
    }
 
    // 8. allocate device memory
    float* d_A;
    float* d_B;
    hipMalloc((void**) &d_A, mem_size_A);
    hipMalloc((void**) &d_B, mem_size_B);
 
    // 9. copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,hipMemcpyHostToDevice);

 
    // 4. allocate host memory for the result C
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* h_C = (float*) malloc(mem_size_C);
 
    // 10. allocate device memory for the result
    float* d_C;
	if(hipMalloc((void**) &d_C, mem_size_C) != hipSuccess){
		printf("Unable to allocate momory on Device");
		getchar();
		return 0;
	}
 
    // 5. perform the calculation
    //    setup execution parameters
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(WC / threadsPerBlock.x, HC / threadsPerBlock.y);
 
	clock_t begin,end;
	begin=clock();
    //   execute the kernel
    matrixMul<<< blocks, threadsPerBlock >>>(d_C, d_A,d_B, WA, WB);
	
    // 11. copy result from device to host
	if(hipMemcpy(h_C, d_C, mem_size_C,hipMemcpyDeviceToHost) !=hipSuccess){
		
		return 0;
	}
 
    // 6. print out the results
    printf("\n\nMatrix C (Results)\n");
    for(int i = 0; i < size_C; i++)
    {
      printf("%3.3f ", h_C[i]);
       if(((i + 1) % WC) == 0)
          printf("\n");
    }
    printf("\n");
 //cout<<"time="<<time<<endl;
	end=clock();

	double time=(double)(end-begin)/(CLOCKS_PER_SEC*1.0);
	
	printf("time=%.6f\n",time);
    // 7. clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    getchar
		();
	return 0;
}