#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

 
#define MAX_THREADS	128 
#define N		512

int*	r_values;
int*	d_values;

void initialize(int *values)
{
	int val=N;
	for(int i=0;i<N;i++)
	{
		value[i]=val;
		val-=1;
	}
}


 // Kernel function
 __global__ static void quicksort(int* values) 
 {
 #define MAX_LEVELS	300

	int pivot, L, R;
	int idx =  threadIdx.x + blockIdx.x * blockDim.x;
	int start[MAX_LEVELS];
	int end[MAX_LEVELS];

	start[idx] = idx;
	end[idx] = N - 1;
	while (idx >= 0) {
		L = start[idx];
		R = end[idx];
		if (L < R) {
			pivot = values[L];
			while (L < R) {
				while (values[R] >= pivot && L < R)
					R--;
				if(L < R)
					values[L++] = values[R];
				while (values[L] < pivot && L < R)
					L++;
				if (L < R)
					values[R--] = values[L];
			}
			values[L] = pivot;
			start[idx + 1] = L + 1;
			end[idx + 1] = end[idx];
			end[idx++] = L;
			if (end[idx] - start[idx] > end[idx - 1] - start[idx - 1]) {
	                        // swap start[idx] and start[idx-1]
        	                int tmp = start[idx];
                	        start[idx] = start[idx - 1];
                        	start[idx - 1] = tmp;

	                        // swap end[idx] and end[idx-1]
        	                tmp = end[idx];
                	        end[idx] = end[idx - 1];
                        	end[idx - 1] = tmp;
	                }

		}
		else
			idx--;
	}
}
 
 // program main
 int main(int argc, char **argv) 
 {
	printf("./quicksort starting with %d numbers...\n", N);
 	unsigned int hTimer;
 	size_t size = N * sizeof(int);
 	
 	// allocate host memory
 	r_values = (int*)malloc(size);
 	
	// allocate device memory
    hipMalloc((void**)&d_values, size) ;

	// allocate threads per block
    const unsigned int cThreadsPerBlock = 128;
                
	

	// initialize data set
    initialize(r_values);

	// copy data to device	
	hipMemcpy(d_values, r_values, size, hipMemcpyHostToDevice);

	printf("Beginning kernel execution...\n");

	
	
	// execute kernel
 	quicksort <<< MAX_THREADS / cThreadsPerBlock, MAX_THREADS / cThreadsPerBlock, cThreadsPerBlock >>> (d_values);
	

 	
 	
	// copy data back to host
	hipMemcpy(r_values, d_values, size, hipMemcpyDeviceToHost);
 	
	// test print
 	for (int i = 0; i < N; i++) 
 	{
 		printf("%d ", r_values[i]);
 	}
 		printf("\n");
		

	// test
    printf("\nTesting results...\n");
   	for (int x = 0; x < N - 1; x++)
   	{
        if (r_values[x] > r_values[x + 1]) 
        {
            printf("Sorting failed.\n");
            break;
       	}
        else
        if (x == N - 2)
               printf("SORTING SUCCESSFUL\n");
    }

	
 	
 	// free memory
	hipFree(d_values) ;
 	free(r_values);
 	
 	
}
