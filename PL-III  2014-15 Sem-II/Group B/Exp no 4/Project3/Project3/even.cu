#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<conio.h>


__global__ void odd(int *arr,int n){
	int i=threadIdx.x;
	int temp;
	if(i%2==1&&i<n-1){
	if(arr[i]>arr[i+1])
	{
	temp=arr[i];
	arr[i]=arr[i+1];
	arr[i+1]=temp;
	}
	}
}

__global__ void even(int *arr,int n){
	int i=threadIdx.x;
	int temp;
	if(i%2==0&&i<n-1){
	if(arr[i]>arr[i+1])
	{
	temp=arr[i];
	arr[i]=arr[i+1];
	arr[i+1]=temp;
	}
	}
}

int main(){
	int SIZE,k,*A,p,j;
	int *d_A;
	
	
	printf("Enter the size of the array\n");
	scanf("%d",&SIZE);
	A=(int *)malloc(SIZE*sizeof(int));
	hipMalloc(&d_A,SIZE*sizeof(int));
	for(k=0;k<SIZE;k++)
		scanf("%d",&A[k]);
		
		
	
	hipMemcpy(d_A,A,SIZE*sizeof(int),hipMemcpyHostToDevice);
	if(SIZE%2==0)
		p=SIZE/2;
	else
		p=SIZE/2+1;
	for(j=0;j<p;j++){
		even<<<2,SIZE>>>(d_A,SIZE);
		if(j!=p-1)
			odd<<<2,SIZE>>>(d_A,SIZE);
		if(j==p-1&&SIZE%2==0)
			odd<<<1,SIZE>>>(d_A,SIZE);
	}
	hipMemcpy(A,d_A,SIZE*sizeof(int),hipMemcpyDeviceToHost);
	for(k=0;k<SIZE;k++)
		printf("%d ",A[k]);
	
	free(A);
	hipFree(d_A);
	
	getch();
	
}